#include <iostream>
#include <hip/hip_runtime.h>
#include <random>
using namespace std;

#define BLOCK_SIZE 32
// 随机初始化
void random_ints(float* a, int n) 
{
    random_device rd;
    mt19937 gen(rd());
    uniform_real_distribution<> dis(0.0, 5.0);

    for (int i = 0; i < n; i++) 
    {
        a[i] = dis(gen);
        // a[i] = 1.0;
    }
}
// 矩阵乘法
__global__ void mat_mul(float *A, float *B, float *C, int M, int N, int K) 
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // 每个线程计算一个C[i][j]
    if (row < M && col < K) 
    {
        float sum = 0;
        for (int i = 0; i < N; i++) 
        {
            sum += A[row * N + i] * B[i * K + col];
        }
        C[row * K + col] = sum;
    }
}

int main() 
{
    // 查看设备属性
    int device;
    hipDeviceProp_t prop;
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);

    cout << "Number of thread blocks: " << prop.maxGridSize[0] << endl;
    cout << "Max threads per block: " << prop.maxThreadsPerBlock << endl;
    
    int M, N, K;
    cin >> M >> N >> K;
    float *A, *B, *C; 
    float *d_A, *d_B, *d_C;

    int size_a = M * N * sizeof(float);
    int size_b = N * K * sizeof(float);
    int size_c = M * K * sizeof(float);

    // 分配设备内存
    hipMalloc((void**)&d_A, size_a);
    hipMalloc((void**)&d_B, size_b);
    hipMalloc((void**)&d_C, size_c);

    // 分配主机内存以及随机初始化
    A = (float*)malloc(size_a);
    random_ints(A, M * N);
    B = (float*)malloc(size_b);
    random_ints(B, N * K);
    C = (float*)malloc(size_c);

    // 将数据从主机拷贝到设备
    hipMemcpy(d_A, A, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size_b, hipMemcpyHostToDevice);
    
    // 设置线程块和网格大小
    int block_y = BLOCK_SIZE;
    int block_x = 512 / BLOCK_SIZE;
    dim3 dimBlock(block_x, block_y);
    dim3 dimGrid((K + block_x - 1) / block_x, (M + block_y - 1) / block_y);
    
    // 记录开始时间
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // 进行矩阵乘法
    mat_mul<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, M, N, K);

    // 记录结束时间
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float time;
    hipEventElapsedTime(&time, start, stop);

    // 销毁事件
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // 将数据传回主机
    hipMemcpy(C, d_C, size_c, hipMemcpyDeviceToHost);
    cout << "Part Result: " << C[2233]  << "  " << C[6044] << "  " << C[M * K - 1] << endl;
    cout << "Used time: " << time << "ms" << endl;

    // 释放内存
    free(A);
    free(B);
    free(C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}