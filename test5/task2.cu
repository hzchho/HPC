#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <random>
using namespace std;

void random_ints(float* a, int n) 
{
    random_device rd;
    mt19937 gen(rd());
    uniform_real_distribution<> dis(0.0, 5.0);

    for (int i = 0; i < n; i++) 
    {
        a[i] = dis(gen);
    }
}

int main()
{
    int M, N, K;
    cin >> M >> N >> K;
    float *A, *B, *C;
    float *d_A, *d_B, *d_C;

    int size_a = M * N * sizeof(float);
    int size_b = N * K * sizeof(float);
    int size_c = M * K * sizeof(float);

    A = (float *)malloc(size_a);
    random_ints(A, M * N);
    B = (float *)malloc(size_b);
    random_ints(B, N * K);
    C = (float *)malloc(size_c);
    
    // 分配设备内存
    hipMalloc((void**)&d_A, size_a);
    hipMalloc((void**)&d_B, size_b);
    hipMalloc((void**)&d_C, size_c);
    
    // 将数据从主机内存复制到设备内存
    hipMemcpy(d_A, A, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size_b, hipMemcpyHostToDevice);

    // 创建cublas句柄
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    // 记录开始时间
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    const float alpha = 1.0f;
    const float beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, K, N, 
                &alpha, d_A, M, d_B, N, &beta, d_C, M);

    // 记录结束时间
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float time;
    hipEventElapsedTime(&time, start, stop);

    // 销毁事件
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // 将数据从设备内存复制到主机内存
    hipMemcpy(C, d_C, size_c, hipMemcpyDeviceToHost);
    cout << "Part Result: " << C[2233]  << "  " << C[6044] << "  " << C[M * K - 1] << endl;
    cout << "Used time: " << time << "ms" << endl;

    hipblasDestroy(handle);
    free(A);
    free(B);
    free(C);

    return 0;
}